#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "phantom.h"
#include "boot/Bootstrapper.cuh"
#include <vector>
#include <cmath>
#include <random>
#include <memory>
#include <tuple>

using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;
using namespace std;

std::vector<complex<double>> generate_random_vector(size_t size) {
    std::vector<complex<double>> result(size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for (size_t i = 0; i < size; ++i) {
        result[i] = complex<double>(dis(gen), dis(gen));
    }
    return result;
}
std::tuple<double, double, size_t> calculateErrorStats(vector<complex<double>> output_vector, vector<complex<double>> standard_vector) {
    if(output_vector.size() != standard_vector.size()){
        throw std::invalid_argument("Input vectors must have the same size");
    }
    if(output_vector.empty()){
        return std::make_tuple(0.0, 0.0, 0);
    }
    double sumAbsError = 0.0;
    double maxError = 0.0;
    size_t maxError_index = 0;
    for(size_t i = 0; i < output_vector.size(); i++){
        double error = abs(output_vector[i] - standard_vector[i]);
        sumAbsError += error;
        if (maxError < error){
            maxError = error;
            maxError_index = i;
        }
    }
    sumAbsError /= output_vector.size();
    return std::make_tuple(sumAbsError, maxError, maxError_index);
}

void main(){
    size_t poly_modulus_degree = 8192;
    const vector<int>& coeff_modulus = {60, 40, 40, 60};
    double scale = pow(2.0, 40);
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(phantom::arith::CoeffModulus::Create(poly_modulus_degree, coeff_modulus));

    PhantomContext context(parms);
    PhantomCKKSEncoder encoder(context);
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    PhantomGaloisKey galois_keys = secret_key.create_galois_keys(context);

    CKKSEvaluator ckks_evaluator(&context, &public_key, &secret_key, &encoder, &relin_keys, &galois_keys, scale);

    int slots = ckks_evaluator.encoder.slot_count();
    vector<complex<double>> input1_vector = generate_random_vector(slots);
    vector<complex<double>> input2_vector = generate_random_vector(slots);
    vector<complex<double>> input3_vector = generate_random_vector(slots);

    PhantomPlaintext plain1, plain2, plain3;
    ckks_evaluator.encoder.encode(input1_vector, scale, plain1);
    ckks_evaluator.encoder.encode(input2_vector, scale, plain2);
    ckks_evaluator.encoder.encode(input3_vector, scale, plain3);

    //add
    PhantomCiphertext cipher1_add, cipher2_add, dest_add;
    ckks_evaluator.encryptor.encrypt(plain1, cipher1_add);
    ckks_evaluator.encryptor.encrypt(plain2, cipher2_add);
    vector<PhantomCiphertext> babyct(3, PhantomCiphertext());
    for (int i = 0; i < 3; i++){
        babyct[i] = dest_add;
    }
    cout <<  "ok" << endl;
}
